#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <vector>
#include <hip/hip_runtime.h>
#include <fstream>
#include <stdio.h>
#include <stdlib.h>
#include <sys/time.h>
#include <time.h>
#include <unistd.h>
#include <cuda/pipeline>
#include <hip/hip_cooperative_groups.h>

#define cudaCheck2(err) (cudaCheck(err, __FILE__, __LINE__))

#define CEIL_DIV(M, N) (((M) + (N)-1) / (N))

void cudaCheck(hipError_t error, const char *file, int line) {
  if (error != hipSuccess) {
    printf("[CUDA ERROR] at file %s:%d:\n%s\n", file, line,
           hipGetErrorString(error));
    exit(EXIT_FAILURE);
  }
};

template <const int BLOCKSIZE>
__global__ void conv2d_shared_mem_block(int M, int N, 
                                       const double *A, double *B) {
  // the output block that we want to compute in this threadblock
  const uint cRow = blockIdx.y;
  const uint cCol = blockIdx.x;
  
  // Need to include cooperative groups for block synchronization
  namespace cg = cooperative_groups;
  auto block = cg::this_thread_block();

  double filter[9] = {-1.0, -1.0, -1.0,
          -1.0, 8.0, -1.0,
          -1.0, -1.0, -1.0};

  // allocate buffer for current block including padding in fast shared mem
  // shared mem is shared between all threads in a block
  __shared__ double As[(BLOCKSIZE + 2) * (BLOCKSIZE + 2)];

  // the inner row & col that we're accessing in this thread
  const uint threadCol = threadIdx.x % BLOCKSIZE;
  const uint threadRow = threadIdx.x / BLOCKSIZE;

  // Set up async pipeline with default 2-stage depth
  auto pipe = cuda::make_pipeline();

  // Each threadblock cooperatively loads the entire block in a single TMA operation
  // This is much more efficient than issuing many small transfers
  
  // Since we need to coordinate across the block, we'll use thread 0 to issue the copy
  if (threadIdx.x == 0) {
    // Acquire a slot in the pipeline
    pipe.producer_acquire();
    
    // Calculate source and destination addresses
    const double* src_ptr = &A[cRow * BLOCKSIZE * (N+2) + cCol * BLOCKSIZE];
    double* dst_ptr = &As[0];
    
    // Create a struct to define the 2D memory layout
    struct MemcpyParams {
      // Size in elements (not bytes)
      size_t rows = BLOCKSIZE + 2;
      size_t cols = BLOCKSIZE + 2;
      
      // Stride in elements (not bytes)
      size_t src_stride = N + 2;
      size_t dst_stride = BLOCKSIZE + 2;
    } params;
    
    // Issue a single 2D copy for the entire tile
    // Each row has (BLOCKSIZE+2) elements, and we copy (BLOCKSIZE+2) rows
    cuda::memcpy_async(dst_ptr, src_ptr, 
                      params.cols * sizeof(double), params.rows,
                      params.dst_stride * sizeof(double),
                      params.src_stride * sizeof(double),
                      pipe);
    
    // Commit the copy operation to the pipeline
    pipe.producer_commit();
  }
  
  // Wait for all memory operations to complete
  // Note that even though only one thread issued the copy, all threads need to wait
  pipe.consumer_wait();
  block.sync();  // Ensure all threads see the loaded data
  
  double tmp = 0.0;
  for (int fi = -1 ; fi < 2; fi++) {
    for (int fj = -1; fj < 2; fj++) { 
      tmp += As[(threadRow + fi + 1) * (BLOCKSIZE + 2) + (threadCol + fj + 1)] * filter[(fi + 1) * 3 + (fj + 1)];
    }
  }

  B[(cRow * BLOCKSIZE + threadRow )* N + cCol * BLOCKSIZE + threadCol] = tmp;

}

void randomize_matrix(double *mat, int N) {
  // NOTICE: Use gettimeofday instead of srand((unsigned)time(NULL)); the time
  // precision is too low and the same random number is generated.
  struct timeval time {};
  gettimeofday(&time, nullptr);
  srand(time.tv_usec);
  for (int i = 0; i < N; i++) {
    double tmp = (double)(rand() % 5) + 0.01 * (rand() % 5);
    tmp = (rand() % 2 == 0) ? tmp : tmp * (-1.);
    mat[i] = tmp;
  }
}

int main(int argc, char **argv) {

  long M=4096, N=4096;

  double *A = nullptr, *B = nullptr, 
        *B_ref = nullptr; // host matrices
  double *dA = nullptr, *dB = nullptr, 
        *dB_ref = nullptr; // device matrices

  A = (double *)malloc(sizeof(double) * (M+2) * (N+2));
  B = (double *)malloc(sizeof(double) * M * N);
  B_ref = (double *)malloc(sizeof(double) * M * N);

  randomize_matrix(A, (M+2) * (N+2));

  cudaCheck2(hipMalloc((void **)&dA, sizeof(double) * (M+2) * (N+2)));
  cudaCheck2(hipMalloc((void **)&dB, sizeof(double) * M * N));
  cudaCheck2(hipMalloc((void **)&dB_ref, sizeof(double) * M * N));

  cudaCheck2(hipMemcpy(dA, A, sizeof(double) * (M+2) * (N+2),
                       hipMemcpyHostToDevice));
  cudaCheck2(hipMemcpy(dB, B, sizeof(double) * M * N,
                       hipMemcpyHostToDevice));
  cudaCheck2(hipMemcpy(dB_ref, B_ref, sizeof(double) * M * N,
                       hipMemcpyHostToDevice));


  // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
  // publishing event tasks in the target stream
  float elapsed_time;
  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);
              
  dim3 gridDim(CEIL_DIV(M, 32), CEIL_DIV(N, 32));
  dim3 blockDim(32 * 32);
  hipFuncSetAttribute(reinterpret_cast<const void*>(conv2d_shared_mem_block<32>),
      hipFuncAttributePreferredSharedMemoryCarveout,
      cudaSharedmemCarveoutMaxShared);

  hipEventRecord(beg);
  for (int j = 0; j < 50; j++) {                       
    conv2d_shared_mem_block<32>
      <<<gridDim, blockDim>>>(M, N, dA, dB);
    hipGetLastError(); // Check for async errors during kernel run      
  }

  hipEventRecord(end);
  hipEventSynchronize(end);
  float elapsedTime;
  hipEventElapsedTime(&elapsedTime, beg, end);
  elapsedTime /= 1000.0; // Convert to seconds
  printf("Elapsed time: %.2f s\n", elapsedTime);

  long flops = 9 * M * N;
  printf(
      "Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS. size: "
      "(%ld).\n",
      elapsedTime / 50,
      (50 * flops * 1e-9) / elapsedTime, M);
  fflush(stdout);

  // Clean up
  free(A);
  free(B);
  free(B_ref);
  cudaCheck2(hipFree(dA));
  cudaCheck2(hipFree(dB));
  cudaCheck2(hipFree(dB_ref));

  return 0;
}
