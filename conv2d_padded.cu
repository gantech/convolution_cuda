#include <cstdio>
#include <cstdlib>
#include <ctime>
#include <fstream>
#include <iostream>
#include <runner_padded.cuh>
#include <vector>

#define cudaCheck(err) (cudaCheck(err, __FILE__, __LINE__))

const std::string errLogFile = "matrixValidationFailure.txt";

int main(int argc, char **argv) {
  if (argc != 2) {
    std::cerr << "Please select a kernel (range 0 - 12, 0 for NVIDIA cuBLAS)"
              << std::endl;
    exit(EXIT_FAILURE);
  }

  // get kernel number
  int kernel_num = std::stoi(argv[1]);
  if (kernel_num < 0 || kernel_num > 20) {
    std::cerr << "Please enter a valid kernel number (0-12)" << std::endl;
    exit(EXIT_FAILURE);
  }

  // get environment variable for device
  int deviceIdx = 0;
  if (getenv("DEVICE") != NULL) {
    deviceIdx = atoi(getenv("DEVICE"));
  }
  cudaCheck(hipSetDevice(deviceIdx));

  printf("Running kernel %d on device %d.\n", kernel_num, deviceIdx);

  // print some device info
  // CudaDeviceInfo();

  // Using cudaEvent for gpu stream timing, cudaEvent is equivalent to
  // publishing event tasks in the target stream
  float elapsed_time;
  hipEvent_t beg, end;
  hipEventCreate(&beg);
  hipEventCreate(&end);

  // cuBLAS FLOPs ceiling is reached at 8192
  std::vector<int> SIZE = {128};

  long m, n, max_size;
  max_size = SIZE[SIZE.size() - 1];
  std::cout << "Max size: " << max_size << std::endl;
  
  double *A = nullptr, *Anonpad = nullptr,
         *B = nullptr, *B_ref = nullptr; // host matrices
  double *dA = nullptr, *dAnonpad = nullptr,
        *dB = nullptr, *dB_ref = nullptr; // device matrices

  A = (double *)malloc(sizeof(double) * (max_size+2) * (max_size+2));
  Anonpad = (double *)malloc(sizeof(double) *max_size * max_size);  
  B = (double *)malloc(sizeof(double) * max_size * max_size);
  B_ref = (double *)malloc(sizeof(double) * max_size * max_size);

  randomize_matrix(A, Anonpad, max_size, max_size);

  cudaCheck(hipMalloc((void **)&dA, sizeof(double) * (max_size+2) * (max_size+2)));
  cudaCheck(hipMalloc((void **)&dAnonpad, sizeof(double) * max_size * max_size));  
  cudaCheck(hipMalloc((void **)&dB, sizeof(double) * max_size * max_size));
  cudaCheck(hipMalloc((void **)&dB_ref, sizeof(double) * max_size * max_size));

  cudaCheck(hipMemcpy(dA, A, sizeof(double) * (max_size+2) * (max_size+2),
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dAnonpad, Anonpad, sizeof(double) * max_size * max_size,
                       hipMemcpyHostToDevice)); 
  cudaCheck(hipMemcpy(dB, B, sizeof(double) * max_size * max_size,
                       hipMemcpyHostToDevice));
  cudaCheck(hipMemcpy(dB_ref, B_ref, sizeof(double) * max_size * max_size,
                       hipMemcpyHostToDevice));

  int repeat_times = 50;
  for (int size : SIZE) {
    m = n = size;

    std::cout << "dimensions(m=n) " << m << std::endl;
    // Verify the correctness of the calculation, and execute it once before the
    // kernel function timing to avoid cold start errors
    if (kernel_num != 0) {
      run_kernel(0, m, n, dAnonpad, dB_ref); // cuDNN
      run_kernel(kernel_num, m, n, dA, dB); // Executes the kernel, modifies the result matrix
      cudaCheck(hipDeviceSynchronize());
      cudaCheck(hipGetLastError()); // Check for async errors during kernel run
      hipMemcpy(B, dB, sizeof(double) * m * n, hipMemcpyDeviceToHost);
      hipMemcpy(B_ref, dB_ref, sizeof(double) * m * n, hipMemcpyDeviceToHost);

      if (!verify_matrix(B_ref, B, m * n)) {
        std::cout
            << "Failed to pass the correctness verification against NVIDIA "
               "cuBLAS."
            << std::endl;
        if (m <= 128) {
          std::cout << " Logging faulty output into " << errLogFile << "\n";
          std::ofstream fs;
          fs.open(errLogFile);
          fs << "Anonpad:\n";
          print_matrix(Anonpad, m, n, fs);
          fs << "B:\n";
          print_matrix(B, m, n, fs);
          fs << "Should:\n";
          print_matrix(B_ref, m, n, fs);
        }
        exit(EXIT_FAILURE);
      }
    }

    if (kernel_num == 0)
      run_kernel(kernel_num, m, n, dAnonpad, dB);
    else {
      elapsed_time = run_kernel(kernel_num, m, n, dA, dB, repeat_times);

      elapsed_time /= 1000.; // Convert to seconds

      long flops = 9 * m * n;
      printf(
          "Average elapsed time: (%7.6f) s, performance: (%7.1f) GFLOPS. size: "
          "(%ld).\n",
          elapsed_time / repeat_times,
          (repeat_times * flops * 1e-9) / elapsed_time, m);
      fflush(stdout);
      // make dB and dB_ref equal again (we modified dB while calling our kernel
      // for benchmarking)
      cudaCheck(hipMemcpy(dB, dB_ref, sizeof(double) * m * n,
                          hipMemcpyDeviceToDevice));
    }
  }

  // Free up CPU and GPU space
  free(A);
  free(B);
  free(B_ref);
  hipFree(dA);
  hipFree(dB);
  hipFree(dB_ref);

  return 0;
};
